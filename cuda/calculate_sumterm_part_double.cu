
#include <hip/hip_runtime.h>
__global__ void calculate_sumterm_part(double2 * Up, double2 * Vpl, const double2 * A_t, const double* SR, const unsigned char* nonzero_midx1234s, const unsigned int N, const unsigned int M, const double SK_factor, const unsigned int NUM_NONZERO, const unsigned int NUM_MODES) {
    unsigned int full_thread_idx = threadIdx.x + blockIdx.x*blockDim.x;

    // Calculate the index
    unsigned int Midx = full_thread_idx / N;
    unsigned int Nidx = full_thread_idx % N;
    unsigned int NM = N*M;

    if (full_thread_idx >= NM) {
        return;
    }

    // Compute the tensors
    for (int i = 0; i < NUM_NONZERO; i++) {
        unsigned int midx1 = nonzero_midx1234s[i*4]-1;
        unsigned int midx2 = nonzero_midx1234s[1+i*4]-1;
        unsigned int midx3 = nonzero_midx1234s[2+i*4]-1;
        unsigned int midx4 = nonzero_midx1234s[3+i*4]-1;

        double a = A_t[Nidx+Midx*N+midx2*NM].x;
        double b = A_t[Nidx+Midx*N+midx2*NM].y;
        double c = A_t[Nidx+Midx*N+midx3*NM].x;
        double d = A_t[Nidx+Midx*N+midx3*NM].y;
        double e = A_t[Nidx+Midx*N+midx4*NM].x;
        double f = A_t[Nidx+Midx*N+midx4*NM].y;

        Up[Nidx+Midx*N+midx1*NM].x = Up[Nidx+Midx*N+midx1*NM].x + SK_factor*SR[i]*(a*c*e-b*d*e+a*d*f+c*b*f);
        Up[Nidx+Midx*N+midx1*NM].y = Up[Nidx+Midx*N+midx1*NM].y + SK_factor*SR[i]*(a*d*e+c*b*e-a*c*f+b*d*f);
        Vpl[Nidx+Midx*N+midx1*NM+midx2*NM*NUM_MODES].x = Vpl[Nidx+Midx*N+midx1*NM+midx2*NM*NUM_MODES].x + SR[i]*(c*e+d*f);
        Vpl[Nidx+Midx*N+midx1*NM+midx2*NM*NUM_MODES].y = Vpl[Nidx+Midx*N+midx1*NM+midx2*NM*NUM_MODES].y + SR[i]*(d*e-c*f);
    }
}
